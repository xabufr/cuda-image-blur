#include "hip/hip_runtime.h"
#include "image.h"
#include "common/cpu_anim.h"
#include <iostream>
#include <time.h>

#define DIM 16
#define MAX_DIM 256

bool add = true;

void cleanup(DataBlock *d) {
	hipFree(d->dev_bitmap);
	hipFree(d->dev_output);
}

__device__ int getPixelIndex(int x, int y, int width) {
	int offset = y * width + x;
	return offset * 4;
}
__global__ void kernel(unsigned char *pixels, unsigned char *output, int width,
		int height, int radius) {
	int x, y;
	x = blockIdx.x * blockDim.x + threadIdx.x;
	y = blockIdx.y * blockDim.y + threadIdx.y;
	if (x < width && y < height) {
		unsigned int averageColor[3] = { 0, 0, 0 };
		int pixelOffset = getPixelIndex(x, y, width);

		int count = 0;

		int currentOffset;
		for (int d_x = -radius; d_x <= radius; ++d_x) {
			int currentX = x + d_x;
			if (currentX < width && currentX >= 0) {
				for (int d_y = -radius; d_y <= radius; ++d_y) {
					int currentY = y + d_y;
					if (currentY < height && currentY >= 0) {
						++count;
						currentOffset = getPixelIndex(currentX, currentY,
								width);
						averageColor[0] += pixels[currentOffset];
						averageColor[1] += pixels[currentOffset + 1];
						averageColor[2] += pixels[currentOffset + 2];
					}
				}
			}
		}
		averageColor[0] /= count;
		averageColor[1] /= count;
		averageColor[2] /= count;

		output[pixelOffset + 0] = averageColor[0];
		output[pixelOffset + 1] = averageColor[1];
		output[pixelOffset + 2] = averageColor[2];
		output[pixelOffset + 3] = 255;
	}
}

__global__ void kernelPixellization(unsigned char *pixels, unsigned char *ouput,
		int width, int height, int radius) {
	__shared__ int moyenne[MAX_DIM][3];
	int x = blockIdx.x * radius;
	int y = blockIdx.y * radius + threadIdx.x;

	__shared__ int finalMoyenne[3];
	__shared__ int count[MAX_DIM];

	for (int i(0); i < 3; ++i) {
		moyenne[threadIdx.x][i] = 0;
	}
	count[threadIdx.x] = 0;
	if (y < height) {
		for (int d_x = 0; d_x < radius; ++d_x) {
			if (d_x + x < width) {
				int offset = getPixelIndex(x + d_x, y, width);
				++count[threadIdx.x];
				for (int i = 0; i < 3; ++i) {
					moyenne[threadIdx.x][i] += pixels[offset + i];
				}
			}
		}
	}

	__syncthreads();

	if (threadIdx.x == 0) {
		for (int i = 0; i < 3; ++i) {
			finalMoyenne[i] = 0;
		}
		int finalCount = 0;
		for (int i = 0; i < radius; ++i) {
			finalCount += count[i];
			for (int j = 0; j < 3; ++j) {
				finalMoyenne[j] += moyenne[i][j];
			}
		}
		for (int i = 0; i < 3; ++i) {
			finalMoyenne[i] /= finalCount;
		}
	}
	__syncthreads();

	if (y < height) {
		for (int d_x = 0; d_x < radius; ++d_x) {
			if (d_x + x < width) {
				int offset = getPixelIndex(x + d_x, y, width);
				for (int i(0); i < 3; ++i) {
					ouput[offset + i] = finalMoyenne[i];
				}
			}
		}
	}
}

void generateFramePixellisation(DataBlock *d, int ticks) {
	if ((ticks % 128) == 0)
		add = !add;

	d->radius += (add ? 1 : -1);
	if (d->radius <= 0)
		d->radius = 1;
	std::cout << d->radius << std::endl;
	hipEventRecord(d->start, 0);
	std::size_t pixelsSize = sizeof(unsigned char) * d->image.width()
			* d->image.height() * 4;
	hipMemcpy(d->dev_bitmap, d->image.pixels(), pixelsSize,
			hipMemcpyHostToDevice);

	kernelPixellization<<<
			dim3((d->image.width() / d->radius) + 1,
					(d->image.height() / d->radius) + 1), d->radius>>>(
			d->dev_bitmap, d->dev_output, d->image.width(), d->image.height(),
			d->radius);
	hipEventRecord(d->stop, 0);
	hipEventSynchronize(d->stop);
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, d->start, d->stop);
	std::cout << "Elapsed time: " << elapsedTime << std::endl;

	hipMemcpy(d->bitmap->get_ptr(), d->dev_output, pixelsSize,
			hipMemcpyDeviceToHost);
}
void generateFrame(DataBlock *d, int ticks) {
	if ((ticks % 10) == 0)
		add = !add;

//if ((ticks % 5) == 0) {
	d->radius = (add) ? d->radius + 1 : d->radius - 1;

	hipEventRecord(d->start, 0);
	std::size_t pixelsSize = sizeof(unsigned char) * d->image.width()
			* d->image.height() * 4;
	hipMemcpy(d->dev_bitmap, d->image.pixels(), pixelsSize,
			hipMemcpyHostToDevice);

	kernel<<<dim3((d->image.width() / 15) + 1, (d->image.height() / 15) + 1),
			dim3(16, 16)>>>(d->dev_bitmap, d->dev_output, d->image.width(),
			d->image.height(), d->radius);
	hipEventRecord(d->stop, 0);
	hipEventSynchronize(d->stop);
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, d->start, d->stop);
	std::cout << "Elapsed time: " << elapsedTime << std::endl;

	hipMemcpy(d->bitmap->get_ptr(), d->dev_output, pixelsSize,
			hipMemcpyDeviceToHost);
//}
}

int main() {
	DataBlock data;
	data.radius = 1;
	data.image.loadFromFile("madame.jpg");
	CPUAnimBitmap bitmap(data.image.width(), data.image.height(), &data);
	data.bitmap = &bitmap;

	std::size_t pixelsSize = sizeof(unsigned char) * data.image.width()
			* data.image.height() * 4;

	hipMalloc(&data.dev_output, pixelsSize);
	hipMalloc(&data.dev_bitmap, pixelsSize);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	bitmap.anim_and_exit((void (*)(void*, int))generateFramePixellisation, (void (*)(void*))cleanup ) ;

	return 0;
}
